#include "hip/hip_runtime.h"
/*Author: Rodrigo Gonçalves de Branco
Date: 12/01/2017
*/

#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>
#include <climits>
#include <stdio.h>
#include <omp.h>
#include <thrust/device_vector.h>
#include "cuda_util.h"
using namespace std;


__global__
void prefixsumJAxis(int* v, int N)
{
	int sqrN = N*N;
	for(int k = blockIdx.x; k < N; k += gridDim.x) {
		for(int i = threadIdx.x; i < N; i += blockDim.x) {
			for(int j = 1; j < N; j++) {
				v[sqrN*k + N*i + j] += v[sqrN*k + N*i + j-1];
			}
		}
	}
}

__global__
void prefixsumKAxis(int* v, int N)
{
	int sqrN = N*N;
	for(int j = blockIdx.x; j < N; j += gridDim.x) {
		for(int i = threadIdx.x; i < N; i += blockDim.x) {
			for(int k = 1; k < N; k++) {
				v[sqrN*k + N*i + j] += v[sqrN*(k-1) + N*i + j];
			}
		}
	}
}

__device__ inline
int row_index( unsigned int i, unsigned int M ){
    double m = M;
    double row = (-2*m - 1 + sqrt( (4*m*(m+1) - 8*(double)i - 7) )) / -2;
    if( row == (double)(int) row ) row -= 1;
    return (unsigned int) row;
}


__device__ inline
int column_index( unsigned int i, unsigned int M ){
    unsigned int row = row_index( i, M);
    return  i - M * row + row*(row+1) / 2;
}

__device__ inline
int maxSubArraySum(int* v, int N, int g, int h, int r, int t) {
	int max_so_far = 0, max_ending_here = 0;


	int sqrN = N*N;

	   for(int i = 0; i < N; i++)
	   {
		int tmp1 = v[sqrN*t + N*i + h];
		int tmp2 = r > 0 ? v[sqrN*(r-1) + N*i + h] : 0;
		int tmp3 = g > 0 ? v[sqrN*t + N*i + (g-1)] : 0;

		//Maybe repeated elements were subtracted. If that is true, we need correct it!
		int tmp4 = r > 0 && g > 0 ? v[sqrN*(r-1) + N*i + (g-1)] : 0 ;

		int temp = tmp1 - tmp2 - tmp3 + tmp4;

		//printf("g:%d h:%d r:%d t:%d => %d - %d - %d + %d = %d\n",g,h,r,t,tmp1,tmp2,tmp3,tmp4,temp);

		max_ending_here = max_ending_here + temp;

	     if(max_ending_here < 0)
		max_ending_here = 0;

	     if(max_so_far < max_ending_here)
		max_so_far = max_ending_here;
	    }

	    return max_so_far;
}

__global__
void computeCghrt(int* v, int N, int * result, int dev_id, int devCount, int computationSize, int totalComp)
{
	int maxsofar = INT_MIN;
	//to cover all R e T index
	//printf("blk:%d thd:%d gridDim:%d blockDim:%d\n",blockIdx.x,threadIdx.x,gridDim.x,blockDim.x);

	extern __shared__ int max_block[];

	if(threadIdx.x == 0)
		max_block[0] = INT_MIN;

	__syncthreads();

	for(int blkstep = 0; blkstep < computationSize; blkstep += gridDim.x) {
		int blkstep_dev = blkstep + dev_id*computationSize;
		int r = row_index(blockIdx.x + blkstep_dev,N);
		int t = column_index(blockIdx.x + blkstep_dev,N);

		if(r >= 0 && t >= 0 && r < N && t < N && r <= t) {			

			//to cover all G e H index
			for(int thdstep = 0; thdstep < totalComp; thdstep += blockDim.x) {
				int g = row_index(threadIdx.x + thdstep,N);
				int h = column_index(threadIdx.x + thdstep,N);

				if(g >= 0 && h >= 0 && g < N && h < N && g <= h) {
					int newmax = maxSubArraySum(v,N,g,h,r,t);
					maxsofar = newmax > maxsofar ? newmax : maxsofar;
					
				}	
			}
		}
	}
	
	//atomicMax(result,maxsofar);
	atomicMax(&max_block[0],maxsofar);

	__syncthreads();

	if(threadIdx.x == 0)
		atomicMax(&result[0],max_block[0]);
}

/*void print(int* v, int N) {
	for(int k = 0; k < N; k++) {
		for(int i = 0; i < N; i++) {
			for(int j = 0; j < N; j++) {
				cout<<v[N*N*k + N*i + j]<<" ";
			}
			cout<<endl;
		}
		cout<<endl;
	}	
}*/

int main() {
	//size of cube
	int N;
	cin>>N;

	//cube representation: O(n^3) of space
	int* cube = (int*)malloc(N*N*N*sizeof(int**));

	//Reading the values
	for(int k = 0; k < N; k++) {
		for(int i = 0; i < N; i++) {
			for(int j = 0; j < N; j++) {
				cin>>cube[N*N*k + N*i + j];
			}
		}
	}

	//cout<<"original:"<<endl;

	//print(cube,N);

	int devCount;
	HANDLE_ERROR( hipGetDeviceCount(&devCount));

	thrust::host_vector<int> max_device(devCount);

	int global_max = -1;

	#pragma omp parallel num_threads(devCount) default(shared)
	{
		
		const int dev_id = omp_get_thread_num();

		HANDLE_ERROR( hipSetDevice(dev_id) );
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, dev_id);

		int* dcube;
		HANDLE_ERROR( hipMalloc( (void**)&dcube,  N*N*N*sizeof(int))); 
		HANDLE_ERROR( hipMemcpy( dcube, cube, N*N*N*sizeof(int),hipMemcpyHostToDevice ) );

		thrust::device_vector<int> d_result(1,INT_MIN);

		int numSMs;
		hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);

		dim3 dimThreads(256);
		dim3 dimBlocks(32*numSMs);

		hipEvent_t start,stop;

		HANDLE_ERROR( hipEventCreate(&start) );
		HANDLE_ERROR( hipEventCreate(&stop) );

		HANDLE_ERROR( hipEventRecord(start, 0) );

		prefixsumJAxis<<<dimBlocks,dimThreads>>>(dcube,N);

		HANDLE_ERROR( hipDeviceSynchronize() );

		//HANDLE_ERROR( hipMemcpy( cube, dcube,N*N*N*sizeof(int),hipMemcpyDeviceToHost));

		//cout<<"first ps:"<<endl;

		//print(cube,N);

		prefixsumKAxis<<<dimBlocks,dimThreads>>>(dcube,N);

		HANDLE_ERROR( hipDeviceSynchronize() );

		//cout<<endl<<"second ps:"<<endl;

		//HANDLE_ERROR( hipMemcpy( cube, dcube,N*N*N*sizeof(int),hipMemcpyDeviceToHost));

		//print(cube,N);

		//cout<<"computation size: "<<N*(N+1)/2<<endl;

		int totalComp = ((N*(N+1))>>1);
		int computationSize = (int)(totalComp/devCount);

		if(totalComp % devCount != 0) {
			computationSize++;
		}

		computeCghrt<<<dimBlocks,dimThreads, sizeof(int)>>>(dcube,N,thrust::raw_pointer_cast(d_result.data()),dev_id,devCount,computationSize,totalComp);

		HANDLE_ERROR( hipDeviceSynchronize() );

		max_device[dev_id] = d_result[0];

		#pragma omp barrier

		#pragma omp single 
		{
			for(int i = 0; i < devCount; i++) {
				if(global_max < max_device[i])
					global_max = max_device[i];
			}
		}

		HANDLE_ERROR( hipEventRecord(stop, 0) );
		HANDLE_ERROR( hipEventSynchronize(start) );
		HANDLE_ERROR( hipEventSynchronize(stop) );

		float elapsedtime;	

		HANDLE_ERROR( hipEventElapsedTime(&elapsedtime, start, stop) );

		//int result;
		//HANDLE_ERROR( hipMemcpy( &result, dresult, sizeof(int),hipMemcpyDeviceToHost));

		hipFree(dcube);
		//hipFree(dresult);

		//cout<<result<<endl;

		//printf("%i %.9f\n",result,elapsedtime);
		//printf("%.9f\n",elapsedtime);

		#pragma omp single
		{
			//printf("\nO resultado e: %d\n",global_max);
			//printf("O tempo foi de: %.9f ms para a mmax2d\n", elapsedtime);			
			printf("%d %.9f\n",global_max,elapsedtime);
			//printf("%.9f\n",elapsedtime);
		}

	}

	free(cube);


	return 0;
}
